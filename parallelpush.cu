#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
parallelpush::parallelpush()
{
};
void parallelpush::init(vector<edge>&extenedges,vector<vector<int>>&relate,ginfo ginf){
	nodenum=ginf.enodesize;
	pnodesize=ginf.pnodesize;
	edges=extenedges;
	W=WD+1;
	h=new int[W*pnodesize*LY];
	v=new int[W*pnodesize*LY];
	mark=new int;
	vector<vector<int>>rawneie(pnodesize,vector<int>());
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			rawneie[s].push_back(i+1);
			rawneie[t].push_back(-(i+1));
		}
	max=0;
	for(int i=0;i<rawneie.size();i++)
		if(rawneie[i].size()>max)max=rawneie[i].size();
	max++;
	neie=new int[pnodesize*max];
	for(int i=0;i<pnodesize;i++)
		{
			for(int j=0;j<max;j++)
			{
				if(j<rawneie[i].size())
					neie[i*max+j]=rawneie[i][j];
				else
					neie[i*max+j]=INT_MAX;
			}
		}
	emark=new int[LY*edges.size()];
	esign=new int[LY*edges.size()];
	st=new int[edges.size()];
	te=new int[edges.size()];
	for(int i=0;i<edges.size();i++)
		{
			st[i]=edges[i].s*W;
			te[i]=edges[i].t*W;
		}
	source=new int[pnodesize];
	ends=new int[pnodesize];
	hipMalloc((void**)&dev_h,LY*W*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_mark,sizeof(int));
	hipMalloc((void**)&dev_v,LY*W*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_neie,max*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_esign,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_emark,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_st,edges.size()*sizeof(int));
	hipMalloc((void**)&dev_ends,pnodesize*sizeof(int));
	hipMalloc((void**)&dev_source,pnodesize*sizeof(int));
	hipMalloc((void**)&dev_te,edges.size()*sizeof(int));
	hipMemcpy(dev_mark,mark,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_neie,neie,max*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_st,st,edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_te,te,edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_ends,ends,pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_source,source,pnodesize*sizeof(int),hipMemcpyHostToDevice);
};
__global__ void push2(int*dev_h,int*dev_v,int* dev_esign,int* dev_emark,int*st,int*te,int*neie,int N,int W,int E,int*mark,int max,int*dev_source,int*dev_ends)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int bi=i%N;
	int value=dev_v[i];
	int node=bi/W;
	if(i>=N*LY||value==0||dev_ends[node]==1||dev_h[i]>2*W+1)return;
	if(bi%W==0&&dev_h[i]>W+1&&dev_source[node]==1){dev_v[i]=0;return;}
	int ly=i/N;
	int off=i%W;
	int h=dev_h[i];
	int b=node*max;
	int minheight=INT_MAX;
	int ebj,nbj,hnbj,eid,seid;
	int flag=0;
	for(int j=0;j<max;j++)
	{
		ebj=neie[b+j];
		if(ebj<INT_MAX&&value>0){
			seid=abs(ebj)-1;
			eid=ly*E+seid;
			nbj=-1;
			bool b1=ebj>0&&dev_esign[eid]>0;
			bool b2=ebj<0&&dev_esign[eid]<0&&(abs(dev_esign[eid])==off);
			if(b1||b2)
			{
				if(ebj>0&&off<W-1)
					nbj=te[seid]+off+1;
				if(ebj<0&&off>0)
					nbj=st[seid]+off-1;
				if(nbj<0)continue;
				nbj+=ly*N;
				hnbj=dev_h[nbj];
				if(dev_emark[eid]==0&&h==hnbj+1)
				{
					dev_emark[eid]++;
					value--;
					*mark=1;
				}
				minheight=min(minheight,hnbj);
			}
		}
		else
			break;
	}
	if(value>0&&minheight<INT_MAX){dev_h[i]=minheight+1;*mark=1;}
};
__global__ void aggregate3(int* dev_esign,int* dev_v,int* dev_emark,int* dev_st,int* dev_te,int*dev_h,int W,int E,int N)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=E*LY)return;
		int s,t;
	int bottom=(i/E)*N;
	int bi=i%E;
	if(dev_emark[i]>0)
	{
		if(dev_esign[i]>0)
		{
			s=dev_st[bi];
			t=dev_te[bi]+1;
		}
		if(dev_esign[i]<0)
		{
			t=dev_st[bi];
			s=dev_te[bi]+1;
		}
		s+=bottom;
		t+=bottom;
		for(int k=0;k<W-1;k++)
			{
				int h1=dev_h[s+k];
				int h2=dev_h[t+k];
				if(dev_v[s+k]>0&&h1==h2+1)
				{
					atomicSub(&dev_v[s+k],1);
					atomicAdd(&dev_v[t+k],1);
					dev_esign[i]=(dev_esign[i]>0)?-(k+t)%W:1;
					break;
				}
			}
			
	}
	dev_emark[i]=0;
};
pair<int,int> parallelpush::prepush(int slen,int tlen,int bw)
{
	for(int i=0;i<LY*edges.size();i++)
		emark[i]=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<edges.size();i++)
			esign[i+k*edges.size()]=1;
	for(int i=0;i<W*LY*pnodesize;i++)
		{
			h[i]=0;
			v[i]=0;
		}
	for(int i=0;i<pnodesize;i++)
	{
		source[i]=0;
		ends[i]=0;
	}
	srand(1);
	int ccc=0;
	for(int i=0;i<pnodesize;i++)
	{
		while(slen>0)
			{
			int j=rand()%pnodesize;
			if(source[j]==0)
				{
					slen--;
					source[j]=1;
					for(int k=0;k<LY;k++)
						{
						v[k*nodenum+j*W]=1;
						}
				}
			}
	}
	for(int i=0;i<pnodesize;i++)
	{
		while(tlen>0)
			{
			int j=rand()%pnodesize;
			if(source[j]==0&&ends[j]==0)
				{
					ends[j]=1;
					tlen--;
				}
			}
	}
	for(int i=0;i<LY*edges.size();i++)
	{
		int ran=rand()%100;
		if(ran<50)
			esign[i]=0;
	}
	hipMemcpy(dev_h,h,LY*W*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_v,v,LY*W*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_esign,esign,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_emark,emark,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_ends,ends,pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_source,source,pnodesize*sizeof(int),hipMemcpyHostToDevice);
	*mark=1;
	int time=0;
	time_t start,end;
	start=clock();
	int flag=0;
	int fl2=1;
	while(*mark!=0)
		{
			if(time%20==0)
				{	*mark=0;
					hipMemcpy(dev_mark,mark,sizeof(int),hipMemcpyHostToDevice);
				}
			
			push2<<<LY*nodenum/WORK_SIZE+1,WORK_SIZE>>>(dev_h,dev_v,dev_esign,dev_emark,dev_st,dev_te,dev_neie,nodenum,W,edges.size(),dev_mark,max,dev_source,dev_ends);
			hipMemcpy(emark,dev_emark,LY*edges.size()*sizeof(int),hipMemcpyDeviceToHost);
			aggregate3<<<LY*edges.size()/WORK_SIZE+1,WORK_SIZE>>>(dev_esign,dev_v,dev_emark,dev_st,dev_te,dev_h,W,edges.size(),W*pnodesize);
			
			if(time%20==0)
				hipMemcpy(mark,dev_mark,sizeof(int),hipMemcpyDeviceToHost);
			time++;
	    }	
	int flow=0;
	hipMemcpy(v,dev_v,LY*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	end=clock();
	for(int i=0;i<W*pnodesize*LY;i++)
			if(v[i]!=0)
				{
					int bi=i%(W*pnodesize);
					if(ends[bi/W]==1)flow+=v[i];
	}
	cout<<"GPU flow is "<<flow<<endl;
	hipMemcpy(esign,dev_esign,LY*edges.size()*sizeof(int),hipMemcpyDeviceToHost);
	vector<int>vesign,vvalue,vends;
	for(int i=0;i>edges.size()*LY;i++)
		vesign.push_back(esign[i]);
	for(int i=0;i<nodenum*LY;i++)
		vvalue.push_back(v[i]);
	for(int i=0;i<pnodesize;i++)
		vends.push_back(ends[i]);
	//dilor->checkhop(0,0,vesign,vvalue,vends);
    cout<<"GPU time is: "<<end-start<<endl;
	return make_pair(flow,end-start);
};

void parallelpush:: dellocate()
{
	/*delete[] h;
	delete[] minarray;
	delete[] v;
	delete[] mark;
	delete[] neie;
	delete[] nein;
	delete[]emark;
	delete[]esign;*/
	/*hipFree(dev_h);
	hipFree(dev_mark);
	hipFree(dev_v);
	hipFree(dev_neie);
	hipFree(dev_nein);
	hipFree(dev_esign);
	hipFree(dev_emark);*/
}
parallelpush:: ~parallelpush(){};



